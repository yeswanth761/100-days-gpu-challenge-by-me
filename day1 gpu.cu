
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*sizeof(float));
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;
    int gridsize=ceil(N/blocksize);
    vectorAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
